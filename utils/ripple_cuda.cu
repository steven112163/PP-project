#include "hip/hip_runtime.h"
#include "../include/ripple_cuda.h"

#define NUM_THREADS 25
#define NUM_STREAMS 8

__global__ void vertex_kernel(float *d_current_state,
                              float *d_next_state,
                              int offset, float damp,
                              int surface_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x, z_in_next_state = blockIdx.y * blockDim.y + threadIdx.y;
    int z = z_in_next_state + offset;
    int surface_stride = 3 * surface_size;
    float new_y = -d_next_state[3 * blockDim.x * z_in_next_state + 3 * x + 1];

    if (x == 0 && z == 0) {
        new_y = (d_current_state[surface_stride * z + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x + 1) + 1]) / 1.5f;
    } else if (x == surface_size - 1 && z == 0) {
        new_y = (d_current_state[surface_stride * z + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x - 1) + 1]) / 1.5f;
    } else if (x == 0 && z == surface_size - 1) {
        new_y = (d_current_state[surface_stride * z + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x + 1) + 1]) / 1.5f;
    } else if (x == surface_size - 1 && z == surface_size - 1) {
        new_y = (d_current_state[surface_stride * z + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x - 1) + 1]) / 1.5;
    } else if (z == 0) {
        new_y = (d_current_state[surface_stride * z + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * z + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x + 1) + 1]) / 2.5f;
    } else if (x == 0) {
        new_y = (d_current_state[surface_stride * z + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x + 1) + 1]) / 2.5f;
    } else if (z == surface_size - 1) {
        new_y = (d_current_state[surface_stride * z + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * z + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x + 1) + 1]) / 2.5f;
    } else if (x == surface_size - 1) {
        new_y = (d_current_state[surface_stride * z + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x - 1) + 1]) / 2.5f;
    } else {
        new_y = (d_current_state[surface_stride * z + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * z + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * x + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z - 1) + 3 * (x + 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x - 1) + 1] +
                 d_current_state[surface_stride * (z + 1) + 3 * (x + 1) + 1]) / 4.0f;
    }

    new_y -= new_y / damp;
    d_next_state[3 * blockDim.x * z_in_next_state + 3 * x] = d_current_state[surface_stride * z + 3 * x];
    d_next_state[3 * blockDim.x * z_in_next_state + 3 * x + 1] = new_y;
    d_next_state[3 * blockDim.x * z_in_next_state + 3 * x + 2] = d_current_state[surface_stride * z + 3 * x + 2];
}

__device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 vec_cross(const float3 &a, const float3 &b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

__device__ float3 vec_normalize(const float3 &a) {
    float length = sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
    return make_float3(a.x / length, a.y / length, a.z / length);
}

__global__ void normal_kernel(float *d_current_state,
                              float *d_normal,
                              int offset,
                              int surface_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int z_in_normal = blockIdx.y * blockDim.y + threadIdx.y;
    int z = z_in_normal + offset;
    int surface_stride = 3 * surface_size;

    float3 point = make_float3(d_current_state[surface_stride * z + 3 * x],
                               d_current_state[surface_stride * z + 3 * x + 1],
                               d_current_state[surface_stride * z + 3 * x + 2]);
    float3 new_normal, neg_z, pos_z, neg_x, pos_x;

    if (x == 0 && z == 0) {
        pos_z = make_float3(d_current_state[surface_stride * (z + 1) + 3 * x],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 2]) - point;
        pos_x = make_float3(d_current_state[surface_stride * z + 3 * (x + 1)],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 2]) - point;
        new_normal = vec_cross(pos_z, pos_x);
    } else if (x == surface_size - 1 && z == 0) {
        pos_z = make_float3(d_current_state[surface_stride * (z + 1) + 3 * x],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 2]) - point;
        neg_x = make_float3(d_current_state[surface_stride * z + 3 * (x - 1)],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 2]) - point;
        new_normal = vec_cross(neg_x, pos_z);
    } else if (x == 0 && z == surface_size - 1) {
        neg_z = make_float3(d_current_state[surface_stride * (z - 1) + 3 * x],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 2]) - point;
        pos_x = make_float3(d_current_state[surface_stride * z + 3 * (x + 1)],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 2]) - point;
        new_normal = vec_cross(pos_x, neg_z);
    } else if (x == surface_size - 1 && z == surface_size - 1) {
        neg_z = make_float3(d_current_state[surface_stride * (z - 1) + 3 * x],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 2]) - point;
        neg_x = make_float3(d_current_state[surface_stride * z + 3 * (x - 1)],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 2]) - point;
        new_normal = vec_cross(neg_z, neg_x);
    } else if (z == 0) {
        pos_z = make_float3(d_current_state[surface_stride * (z + 1) + 3 * x],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 2]) - point;
        neg_x = make_float3(d_current_state[surface_stride * z + 3 * (x - 1)],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 2]) - point;
        pos_x = make_float3(d_current_state[surface_stride * z + 3 * (x + 1)],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 2]) - point;
        new_normal = vec_cross(neg_x, pos_z) + vec_cross(pos_z, pos_x);
    } else if (x == 0) {
        neg_z = make_float3(d_current_state[surface_stride * (z - 1) + 3 * x],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 2]) - point;
        pos_z = make_float3(d_current_state[surface_stride * (z + 1) + 3 * x],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 2]) - point;
        pos_x = make_float3(d_current_state[surface_stride * z + 3 * (x + 1)],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 2]) - point;
        new_normal = vec_cross(pos_z, pos_x) + vec_cross(pos_x, neg_z);
    } else if (z == surface_size - 1) {
        neg_z = make_float3(d_current_state[surface_stride * (z - 1) + 3 * x],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 2]) - point;
        neg_x = make_float3(d_current_state[surface_stride * z + 3 * (x - 1)],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 2]) - point;
        pos_x = make_float3(d_current_state[surface_stride * z + 3 * (x + 1)],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 2]) - point;
        new_normal = vec_cross(pos_x, neg_z) + vec_cross(neg_z, neg_x);
    } else if (x == surface_size - 1) {
        neg_z = make_float3(d_current_state[surface_stride * (z - 1) + 3 * x],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 2]) - point;
        pos_z = make_float3(d_current_state[surface_stride * (z + 1) + 3 * x],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 2]) - point;
        neg_x = make_float3(d_current_state[surface_stride * z + 3 * (x - 1)],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 2]) - point;
        new_normal = vec_cross(neg_z, neg_x) + vec_cross(neg_x, pos_z);
    } else {
        neg_z = make_float3(d_current_state[surface_stride * (z - 1) + 3 * x],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z - 1) + 3 * x + 2]) - point;
        pos_z = make_float3(d_current_state[surface_stride * (z + 1) + 3 * x],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 1],
                            d_current_state[surface_stride * (z + 1) + 3 * x + 2]) - point;
        neg_x = make_float3(d_current_state[surface_stride * z + 3 * (x - 1)],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x - 1) + 2]) - point;
        pos_x = make_float3(d_current_state[surface_stride * z + 3 * (x + 1)],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 1],
                            d_current_state[surface_stride * z + 3 * (x + 1) + 2]) - point;
        new_normal = vec_cross(neg_z, neg_x) + vec_cross(neg_x, pos_z) +
                     vec_cross(pos_z, pos_x) + vec_cross(pos_x, neg_z);
    }

    new_normal = vec_normalize(new_normal);
    d_normal[3 * blockDim.x * z + 3 * x] = new_normal.x;
    d_normal[3 * blockDim.x * z + 3 * x + 1] = new_normal.y;
    d_normal[3 * blockDim.x * z + 3 * x + 2] = new_normal.z;
}

void ripple_cuda(Surface *surface, int &state, int &dampI) {
    const int surface_size = surface->get_surface_size();
    const int vertices_size = surface->get_vertices_size() * sizeof(float);
    float damp = float(dampI);

    // Compute grid step and size
    int grid_step = surface_size / NUM_STREAMS;
    int grid_size = vertices_size / NUM_STREAMS;

    // Allocate device memory for vertices
    float *d_current_state, *d_next_state;
    hipMalloc(&d_current_state, vertices_size);
    hipMalloc(&d_next_state, grid_size);
    hipHostRegister(surface->vertices[0], vertices_size, hipHostRegisterPortable);
    hipHostRegister(surface->vertices[1], vertices_size, hipHostRegisterPortable);

    // Copy current state to device
    hipMemcpy(d_current_state, surface->vertices[state], vertices_size, hipMemcpyHostToDevice);

    // Create streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamCreate(&streams[i]);

    // Start kernels
    dim3 block(NUM_THREADS, NUM_THREADS);
    dim3 grid(surface_size / NUM_THREADS, grid_step / NUM_THREADS);
    int offset = 0;
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipMemcpyAsync(d_next_state + vertices_size * offset,
                        surface->vertices[1 - state] + vertices_size * offset,
                        grid_size, hipMemcpyHostToDevice, streams[i]);
        vertex_kernel<<<grid, block, 0, streams[i]>>>(
                d_current_state,
                d_next_state,
                offset, damp,
                surface_size
        );
        hipMemcpyAsync(surface->vertices[1 - state] + vertices_size * offset,
                        d_next_state + vertices_size * offset,
                        grid_size, hipMemcpyDeviceToHost, streams[i]);
        offset += grid_step;
    }

    // Free vertices space
    hipDeviceSynchronize();
    hipHostUnregister(surface->vertices[state]);
    hipFree(d_next_state);

    // Flip state
    state = 1 - state;

    // Allocate device memory for normals
    float *d_normal;
    hipMalloc(&d_normal, grid_size);
    hipHostRegister(surface->normals.data(), vertices_size, hipHostRegisterPortable);

    // Copy current state to device
    hipMemcpy(d_current_state, surface->vertices[state], vertices_size, hipMemcpyHostToDevice);

    // Start kernels
    offset = 0;
    for (int i = 0; i < NUM_STREAMS; i++) {
        normal_kernel<<<grid, block, 0, streams[i]>>>(
                d_current_state,
                d_normal,
                offset,
                surface_size
        );
        hipMemcpyAsync(surface->normals.data() + vertices_size * offset,
                        d_normal + vertices_size * offset,
                        grid_size, hipMemcpyDeviceToHost, streams[i]);
        offset += grid_step;
    }

    // Free space
    hipDeviceSynchronize();
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamDestroy(streams[i]);
    hipHostUnregister(surface->vertices[state]);
    hipHostUnregister(surface->normals.data());
    hipFree(d_next_state);
    hipFree(d_normal);
}